#include "hip/hip_runtime.h"
//
// Created by ascdc on 2021-11-29.
//



#include "documentParser.cuh"





/**
 * Check if every character in string is equal to ch
 * @param devInputStr
 * @return output
 * @param ch
 *
 */
__global__ void tagChar(array<charType> devInputStr, array<size_t> output, charType ch){
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx >= devInputStr.size)
        return;
    if(devInputStr.ptr[idx] == ch){
        output.ptr[idx] = true;
    }
}

/**
 * create document token list and line info
 * @param scanSpace
 * @param scanBreak
 * @param str
 * @return token
 * @return document
 * @param idx
 */
__host__ __device__ void  writeTokenData(array<size_t> scanSpace, array<size_t> scanBreak, array<charType > str, array<wordAndPartOfSpeechPair> token, array<documentSentenceNode> document, size_t idx){
    if(idx >= scanSpace.size)
        return;
    size_t myTokenId = scanSpace.ptr[idx] / 2;
    bool isPOS = scanSpace.ptr[idx] & 1;
    if(!isPOS){
        if(idx == 0){
            token.ptr[myTokenId].begin = 0;
            token.ptr[myTokenId].id = myTokenId;
        }else if(scanSpace.ptr[idx] != scanSpace.ptr[idx - 1]){
            //和左邊的不一樣
            token.ptr[myTokenId].begin = idx;
            token.ptr[myTokenId].id = myTokenId;

        }
        if(scanSpace.ptr[idx] != scanSpace.ptr[idx + 1]){
            //和右邊的不一樣
            token.ptr[myTokenId].end = idx;
            token.ptr[myTokenId].id = myTokenId;
        }


    }else if(str.ptr[idx] == L'('){
        uint16_t num = 0;
        auto it = idx + 1;
        while(str.ptr[it] != L')'){
            num *= 10;
            num += str.ptr[it] - L'0';
            it++;
        }
        token.ptr[myTokenId].partOfSpeech = num;
    }

    size_t mySentenceId = scanBreak.ptr[idx];
    if(idx == 0){
        document.ptr[mySentenceId].id = mySentenceId;
        document.ptr[mySentenceId].nodeBegin = myTokenId;
    }else if(idx == scanBreak.size - 1){
        document.ptr[mySentenceId].id = mySentenceId;
        document.ptr[mySentenceId].nodeEnd = myTokenId + 1;
    }else{
        if(scanBreak.ptr[idx] != scanBreak.ptr[idx - 1]){
            //和左邊的不一樣
            document.ptr[mySentenceId].nodeBegin = myTokenId;
            document.ptr[mySentenceId].id = mySentenceId;

        }

        if(scanBreak.ptr[idx] != scanBreak.ptr[idx + 1]){
            //和右邊的不一樣
            document.ptr[mySentenceId].nodeEnd = myTokenId + 1;
            document.ptr[mySentenceId].id = mySentenceId;
        }
    }

}

__global__ void writeTokenData(array<size_t> scanSpace, array<size_t> scanBreak, array<charType > str, array<wordAndPartOfSpeechPair> token, array<documentSentenceNode> document){
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    writeTokenData(scanSpace, scanBreak, str, token, document, idx);

}

/**
 * convert input string to  documentToken
 * @param input
 * @return documentToken(devive ptr)
 */
auto getDocumentToken(std::wstring &input) -> documentToken{
    array<charType> devInput;
    auto error = hipMalloc(reinterpret_cast<void **>(&(devInput.ptr)), sizeof(charType) * input.size());
    devInput.size = input.size();
    if(error != hipSuccess){
        throw __FILE__ + std::to_string(__LINE__) + __func__  + hipGetErrorName(error)+ "\n";
    }
    error = hipMemcpy(devInput.ptr, input.c_str(), sizeof(charType) * devInput.size, hipMemcpyHostToDevice);
    if(error != hipSuccess){
        throw __FILE__ + std::to_string(__LINE__) + __func__  + hipGetErrorName(error)+ "\n";
    }

    array<size_t> isSpace;
    array<size_t> isBreak;
    isSpace.size = devInput.size;
    isBreak.size = devInput.size;
    error = hipMalloc(reinterpret_cast<void **>(&(isSpace.ptr)), isSpace.size * sizeof(size_t));
    if(error != hipSuccess){
        throw __FILE__ + std::to_string(__LINE__) + __func__  + hipGetErrorName(error)+ "\n";
    }
    error = hipMalloc(reinterpret_cast<void **>(&(isBreak.ptr)), isBreak.size * sizeof(size_t));
    if(error != hipSuccess){
        throw __FILE__ + std::to_string(__LINE__) + __func__  + hipGetErrorName(error)+ "\n";
    }

    tagChar<<<devInput.size / 512 + 1, 512>>>(devInput, isSpace, L' ');
    hipDeviceSynchronize();
    tagChar<<<devInput.size / 512 + 1, 512>>>(devInput, isBreak, L'\n');
    hipDeviceSynchronize();
    tagChar<<<devInput.size / 512 + 1, 512>>>(devInput, isSpace, L'\n');
    hipDeviceSynchronize();
    error = hipGetLastError();

    if(error != hipSuccess){
        throw __FILE__ + std::to_string(__LINE__) + __func__  + hipGetErrorName(error)+ "\n";
    }

    array<size_t> scanSpace;
    array<size_t> scanBreak;
    scanSpace.size = devInput.size;
    scanBreak.size = devInput.size;

    error = hipMalloc(reinterpret_cast<void **>(&(scanSpace.ptr)), scanSpace.size * sizeof(size_t));
    if(error != hipSuccess){
        std::cout << scanSpace.size  << std::endl;
        std::cout << scanSpace.size * sizeof(size_t) << std::endl;
        throw __FILE__ + std::to_string(__LINE__) + __func__  + hipGetErrorName(error)+ "\n";
    }
    error = hipMalloc(reinterpret_cast<void **>(&(scanBreak.ptr)), scanBreak.size * sizeof(size_t));
    if(error != hipSuccess){
        throw __FILE__ + std::to_string(__LINE__) + __func__  + hipGetErrorName(error)+ "\n";
    }
    thrust::exclusive_scan(thrust::device, isSpace.ptr, isSpace.ptr + isSpace.size, scanSpace.ptr);
    thrust::exclusive_scan(thrust::device, isBreak.ptr, isBreak.ptr + isBreak.size, scanBreak.ptr);



    array<wordAndPartOfSpeechPair> token;
    array<documentSentenceNode> document;
    error = hipMemcpy(&(token.size), scanSpace.ptr + scanSpace.size - 1, sizeof(size_t), hipMemcpyDeviceToHost);
    if(error != hipSuccess){
        throw __FILE__ + std::to_string(__LINE__) + __func__  + hipGetErrorName(error)+ "\n";
    }
    error = hipMemcpy(&(document.size), scanBreak.ptr + scanBreak.size - 1, sizeof(size_t), hipMemcpyDeviceToHost);
    if(error != hipSuccess){
        throw __FILE__ + std::to_string(__LINE__) + __func__  + hipGetErrorName(error)+ "\n";
    }
    token.size += 1;
    token.size /= 2;
    document.size++;
    error = hipMalloc(reinterpret_cast<void **>(&(token.ptr)), sizeof(wordAndPartOfSpeechPair) * token.size);
    if(error != hipSuccess){
        throw __FILE__ + std::to_string(__LINE__) + __func__  + hipGetErrorName(error)+ "\n";
    }
    error = hipMalloc(reinterpret_cast<void **>(&(document.ptr)), sizeof(documentSentenceNode) * document.size);
    if(error != hipSuccess){
        throw __FILE__ + std::to_string(__LINE__) + __func__  + hipGetErrorName(error)+ "\n";
    }

    writeTokenData<<<devInput.size / 1024 + 1, 1024>>>(scanSpace, scanBreak, devInput, token, document);
    /*
    {
        auto hostScanSpace = scanSpace;
        auto hostScanBreak = scanBreak;
        auto hostToken = token;
        auto hostDocument = document;
        auto hostInput = devInput;
        hostScanSpace.ptr = new size_t[hostScanSpace.size];
        hostScanBreak.ptr = new size_t[hostScanBreak.size];
        hostInput.ptr = new charType[hostInput.size];
        hostToken.ptr = new wordAndPartOfSpeechPair[hostToken.size];
        hostDocument.ptr = new documentSentenceNode[hostDocument.size];
        hipMemcpy(hostScanSpace.ptr, scanSpace.ptr, scanSpace.size * sizeof(size_t), hipMemcpyDeviceToHost);
        hipMemcpy(hostScanBreak.ptr, scanBreak.ptr, scanBreak.size * sizeof(size_t), hipMemcpyDeviceToHost);
        hipMemcpy(hostInput.ptr, devInput.ptr, devInput.size * sizeof(charType), hipMemcpyDeviceToHost);
        for(auto idx = 0 ; idx < devInput.size ; idx++){
            writeTokenData(hostScanSpace, hostScanBreak, hostInput, hostToken, hostDocument, idx);
        }

    }
    */
    hipDeviceSynchronize();
    error = hipGetLastError();
    if(error != hipSuccess){
        throw __FILE__ + std::to_string(__LINE__) + __func__  + hipGetErrorName(error)+ "\n";
    }

    /*{
        array<wordAndPartOfSpeechPair> hostToken;
        array<documentSentenceNode> hostDoc;
        hostToken.size = token.size;
        hostDoc.size = document.size;
        hostToken.ptr = new wordAndPartOfSpeechPair[hostToken.size];
        hostDoc.ptr = new documentSentenceNode[hostDoc.size];
        hipMemcpy(hostToken.ptr, token.ptr, token.size * sizeof(wordAndPartOfSpeechPair), hipMemcpyDeviceToHost);
        hipMemcpy(hostDoc.ptr, document.ptr, document.size * sizeof(documentSentenceNode), hipMemcpyDeviceToHost);

        for(auto it = 0 ; it < hostToken.size ; it++){
            std::wcout << "id: " << hostToken.ptr[it].id << std::endl;
            std::wcout << "pos: " << hostToken.ptr[it].partOfSpeech << std::endl;
            std::wcout << "begin: " << hostToken.ptr[it].begin <<" end: " << hostToken.ptr[it].end << std::endl;
            std::wcout << input.substr(hostToken.ptr[it].begin, hostToken.ptr[it].end - hostToken.ptr[it].begin) << std::endl;
            std::wcout << "@@@@@@@@@@@@\n";
        }
        for(auto it = 0 ; it < document.size ; it++){
            std::wcout << "docId: " << hostDoc.ptr[it].id << std::endl;
            std::wcout << "from: " << hostDoc.ptr[it].nodeBegin << " to: " << hostDoc.ptr[it].nodeEnd << std::endl;
            std::wcout << "@@@@@@@@@@@@\n";

        }
    }*/





    hipFree(isSpace.ptr);
    hipFree(isBreak.ptr);
    hipFree(scanSpace.ptr);
    hipFree(scanBreak.ptr);
    return {
        token,
        document,
        devInput
    };
}