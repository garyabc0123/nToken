//
// Created by ascdc on 2021-11-26.
//
#include "operatorParser.cuh"


/**
 * 詞法分析（英語：lexical analysis）是計算機科學中將字符序列轉換為標記（token）序列的過程。進行詞法分析的程序或者函數叫作詞法分析器
 * （lexical analyzer，簡稱lexer），也叫掃描器（scanner）。詞法分析器一般以函數的形式存在，供語法分析器調用。
 * @param input
 * @return token stream
 */
auto lecicalAnalyzer(std::wstring input) -> std::vector<symbolTokenStream>{

    std::vector<symbolTokenStream> ret;
    std::deque<wchar_t> buffer;
    size_t id = 0;
    for (size_t it = 0; it < input.size() ; it++){
        switch (input[it]) {
            case L'\\':
                buffer.push_back(input[it + 1]);//TODO have bug ex: \% {}
                it++;
                break;
            case L'$':
            case L'%':
            case L'|':
            case L'!':
            case L'^':
            case L'[':
            case L']':
            case L'{':
            case L'}':
            {
                if(!buffer.empty()){
                    ret.push_back(symbolTokenStream{id, symbolTable::str, std::wstring(buffer.begin(), buffer.end())});
                    id++;
                    buffer.clear();

                }
                wchar_t ch = input.at(it);
                ch += 0x100000;
                ret.push_back(symbolTokenStream{id, symbolTable::str, std::wstring(1, ch)});
                //offset to unicode "Private Use Area-B"
                id++;
                break;
            }

            case L' ':
            case L'\n':
            case L'\t':
                if(!buffer.empty()){
                    ret.push_back(symbolTokenStream{id, symbolTable::str, std::wstring(buffer.begin(), buffer.end())});
                    id++;
                    buffer.clear();

                }
                break;
            default:
                buffer.push_back(input[it]);
        }
    };
    if(!buffer.empty()){
        ret.push_back(symbolTokenStream{id, symbolTable::str, std::wstring(buffer.begin(), buffer.end())});
    }

    //0x100000: offset symbol to Unicode Private Use Area-B
    for(auto it = ret.begin() ; it != ret.end() ; it++){
        if(it->type == symbolTable::str && it->str.size() == 1 && it->str.front() > 0x100000){
            switch (it->str.front() - 0x100000) {
                case L'$' :
                    it->type = symbolTable::dollarSign;
                    it->str.clear();
                    break;
                case L'%' :
                    it->type = symbolTable::percentSign;
                    it->str.clear();
                    break;
                case L'|' :
                    it->type = symbolTable::verticalBar;
                    it->str.clear();
                    break;
                case L'!' :
                    it->type = symbolTable::exclamationMark;
                    it->str.clear();
                    break;
                case L'^' :
                    it->type = symbolTable::caret;
                    it->str.clear();
                    break;
                case L'[' :
                    it->type = symbolTable::squareBracketLeft;
                    it->str.clear();
                    break;
                case L']' :
                    it->type = symbolTable::squareBracketRight;
                    it->str.clear();
                    break;
                case L'{' :
                    it->type = symbolTable::curlyBracketLeft;
                    it->str.clear();
                    break;
                case L'}' :
                    it->type = symbolTable::curlyBracketRight;
                    it->str.clear();
                    break;
                default:
                    //do nothing
                    break;
            }


        }

    }
    return ret;

}
#ifdef NOONOD
auto infixToPrefix(std::vector<symbolTokenStream> input) -> std::vector<symbolTokenStream>{
    std::stack<symbolTokenStream> stack;
    std::deque<symbolTokenStream> output;
    for(int64_t it = input.size() - 1 ; it >= 0 ; it--){
        switch (input[it].type) {
            case symbolTable::str:
                output.push_back(input[it]);
                break;
            case symbolTable::curlyBracketRight:
                stack.push(input[it]);
                break;
            case symbolTable::curlyBracketLeft:
                for (;!stack.empty();){
                    auto temp = stack.top();
                    stack.pop();
                    if(temp.type == symbolTable::curlyBracketRight){
                        break;
                    }else{
                        output.push_front(temp);
                    }
                }
                break;
            case symbolTable::verticalBar:
            case symbolTable::caret:
                if(stack.empty()){
                    stack.push(input[it]);
                }else{
                    for(;!stack.empty();){
                        auto temp = stack.top();
                        if(temp.type == symbolTable::curlyBracketRight){
                            break;
                        }else if(operatorPriority(temp.type) < operatorPriority(input[it].type)){
                            output.push_front(temp);
                            stack.pop();
                        }else{
                            break;
                        }
                    }
                    stack.push(input[it]);
                }
                break;
            case symbolTable::percentSign:
            case symbolTable::exclamationMark:
            case symbolTable::dollarSign:
            default:
                output.push_front(input[it]);
                break;
        }
    }
    for(;!stack.empty();){
        auto temp = stack.top();
        output.push_front(temp);
        stack.pop();
    }
    return std::vector<symbolTokenStream>(output.begin(), output.end());
}
#endif
auto infixToPrefix(std::vector<symbolTokenStream> input) -> std::vector<symbolTokenStream>{
    std::stack<symbolTokenStream> stack;
    std::deque<symbolTokenStream> output;
    for(size_t it = input.size() - 1;  static_cast<size_t>(it) !=  static_cast<size_t>(-1) ; it--){
        switch (input[it].type) {
            case symbolTable::curlyBracketRight:
                stack.push(input[it]);
                break;
            case symbolTable::curlyBracketLeft:
                while(!stack.empty()){
                    if(stack.top().type == symbolTable::curlyBracketRight){
                        stack.pop();
                        break;
                    }
                    output.push_back(stack.top());
                    stack.pop();
                }
            default:
            {
                if(operatorPriority(input[it].type) < 100){
                    while(!stack.empty()){
                        if(stack.top().type == symbolTable::curlyBracketRight || operatorPriority(input[it].type) <
                                                                                         operatorPriority(stack.top().type)){
                            break;
                        }
                        output.push_back(stack.top());
                        stack.pop();
                    }
                    stack.push(input[it]);
                }else{
                    output.push_back(input[it]);
                }
            }
        }
    }

    while(!stack.empty()){
        output.push_back(stack.top());
        stack.pop();
    }
    std::reverse(output.begin(), output.end());
    return std::vector(output.begin(), output.end());
}


auto prefixToParseTree(std::vector<symbolTokenStream> &input, size_t begin, size_t size, parseTree * me) -> size_t{
    size_t retNext;
    for(size_t it = begin ; it < begin + size && it < input.size() ; it++){
        switch (input[it].type) {
            case symbolTable::dollarSign:
            case symbolTable::percentSign:
            case symbolTable::exclamationMark:
                me->token = input[it];
                me->left = new parseTree;
                it = prefixToParseTree(input, it+1, 1, me->left);
                retNext = it;
                break;
            case symbolTable::verticalBar:
            case symbolTable::caret:
                me->token = input[it];
                me->left = new parseTree;
                me->right = new parseTree;
                it = prefixToParseTree(input, it+1, 1, me->left);
                it = prefixToParseTree(input, it+1, 1, me->right);
                retNext = it;
                break;
            default:
                me->token = input[it];
                retNext = it;
                break;
        }
    }
    return retNext;
}


auto tokenStream2Tree(std::vector<symbolTokenStream> token) -> parseTree *{
    token = infixToPrefix(token);
    parseTree * treeRoot = new parseTree;
    prefixToParseTree(token, 0, token.size(), treeRoot);
    return treeRoot;
}
auto tokenStream2TreeInArray(std::vector<symbolTokenStream> token) -> parseTreeInArray{
    token = infixToPrefix(token);

    std::wstring charArray;
    std::vector<parseTreeInArrayNode> tArray(token.size());

    for(size_t i = 0 ; i < token.size() ; i++){
        tArray[i].type = token[i].type;
        tArray[i].tokenId = token[i].id;
        tArray[i].strInArrayBeginId = charArray.size();
        if(token[i].type == symbolTable::str){
            charArray += token[i].str;
        }
        tArray[i].strInArrayEndId = charArray.size();
    }
    parseTreeInArray ret;
    hipMallocManaged(reinterpret_cast<void **>(&(ret.nodeList)), sizeof(parseTreeInArrayNode) * token.size());
    hipMallocManaged(reinterpret_cast<void **>(&(ret.strArray)), sizeof(charType) * charArray.size());
    memcpy(ret.nodeList, tArray.data(), sizeof(parseTreeInArrayNode) * tArray.size());
    memcpy(ret.strArray, charArray.c_str(), sizeof(charType) * charArray.size());
    ret.nodeListSize = tArray.size();
    ret.strArraySize = charArray.size();



    return ret;

}


/**
 * 語法分析（英語：syntactic analysis，也叫 parsing）是根據某種給定的形式文法對由單詞序列（如英語單詞序列）構成的輸入文字進行分析並確定其語
 * 法結構的一種過程。
 * @param token
 * @return parseTree
 * @return dist
 */
auto syntaxDirectedTranslator(std::vector<symbolTokenStream> token) -> std::tuple<parseTreeInArray *, distList, size_t >{
    int nowState = 0;
    int curlyBegin, curlyEnd = 0;
    int squarBegin, squareEnd = 0;
    std::stack<symbolTable> stack;
    std::vector<parseTreeInArray > computeTupleTree;
    std::vector<size_t> dist;

    for(size_t it = 0 ; it < token.size() ; it++){
        switch (nowState){
            case 0:
                if(token[it].type == symbolTable::curlyBracketLeft){
                    curlyBegin = it;
                    nowState++;
                    stack.push(symbolTable::curlyBracketLeft);
                }
                break;
            case 1:
                if(token[it].type == symbolTable::curlyBracketLeft){
                    stack.push(symbolTable::curlyBracketLeft);
                }else if(token[it].type == symbolTable::curlyBracketRight){
                    if(stack.empty()){
                        throw "Synatex Error, loss {";
                    }else{
                        stack.pop();
                        if(stack.empty()){
                            curlyEnd = it;
                            nowState++;
                        }
                    }
                }
                break;
            case 2:
                if(token[it].type == symbolTable::squareBracketLeft){
                    squarBegin = it;
                    nowState++;
                }
                break;
            case 3:
                if(token[it].type == symbolTable::squareBracketRight){
                    squareEnd = it;
                    nowState = 0;
                    auto computeTree = tokenStream2TreeInArray(std::vector<symbolTokenStream>(token.begin()+curlyBegin+1, token.begin()+curlyEnd));
                    std::wstring distStr(token[squarBegin + 1].str);
                    size_t distThis = std::stoi(distStr) + 1;
                    dist.push_back(distThis);
                    computeTupleTree.push_back(computeTree);
                }
        }


    }
    parseTreeInArray * tempComputeTupleTree ;//= new parseTree*[computeTupleTree.size()];
    hipError_t error;
    error = hipMallocManaged(reinterpret_cast<void **>(&tempComputeTupleTree), sizeof(parseTreeInArray) * computeTupleTree.size());
    if(error != hipSuccess){
        throw __FILE__ + std::to_string(__LINE__) + __func__  + hipGetErrorName(error)+ "\n";
    }
    distList tempDistList;// = new size_t[computeTupleTree.size()];
    error = hipMallocManaged(reinterpret_cast<void **>(&tempDistList), sizeof(size_t) * computeTupleTree.size());
    if(error != hipSuccess){
        throw __FILE__ + std::to_string(__LINE__) + __func__  + hipGetErrorName(error)+ "\n";
    }
    std::copy(computeTupleTree.begin(), computeTupleTree.end(), tempComputeTupleTree);
    std::copy(dist.begin(), dist.end(), tempDistList);
    return std::tuple<parseTreeInArray *, distList, size_t >{tempComputeTupleTree, tempDistList, computeTupleTree.size()};

}


auto compiler(std::wstring searchKey) -> std::tuple<parseTreeInArray *, distList, size_t >{
    auto token = lecicalAnalyzer(searchKey);
    return syntaxDirectedTranslator(token);

}

auto __host__ __device__ operatorPriority(symbolTable in) -> int{
    switch (in) {
        case symbolTable::squareBracketLeft:
        case symbolTable::squareBracketRight:
        case symbolTable::curlyBracketLeft:
        case symbolTable::curlyBracketRight:
                //[ ] { }
                return 3;
        case symbolTable::dollarSign:
        case symbolTable::percentSign:
                //$ %
                return 4;
        case symbolTable::exclamationMark:
                //!
                return 5;
        case symbolTable::caret:
                //^
                return 6;
        case symbolTable::verticalBar:
                //|
                return 7;
        case symbolTable::str:
        case symbolTable::boolean:
            return 100;
        case symbolTable::null:
            return 0;
        default:
            return 0;
        }
}

std::wostream& operator<<(std::wostream& out, const symbolTable value){
    static std::map<symbolTable, std::string> strings;
    if (strings.size() == 0){

#define INSERT_ELEMENT(p) strings[p] = #p
        INSERT_ELEMENT(symbolTable::null);
        INSERT_ELEMENT(symbolTable::dollarSign);
        INSERT_ELEMENT(symbolTable::percentSign);
        INSERT_ELEMENT(symbolTable::verticalBar);
        INSERT_ELEMENT(symbolTable::exclamationMark);
        INSERT_ELEMENT(symbolTable::caret);
        INSERT_ELEMENT(symbolTable::squareBracketLeft);
        INSERT_ELEMENT(symbolTable::squareBracketRight);
        INSERT_ELEMENT(symbolTable::curlyBracketLeft);
        INSERT_ELEMENT(symbolTable::curlyBracketRight);
        INSERT_ELEMENT(symbolTable::str);
        INSERT_ELEMENT(symbolTable::boolean);

#undef INSERT_ELEMENT
    }
    std::wstring temp(strings[value].begin(), strings[value].end());
    return out << temp;
}